//
//  dc3.cu
//  dc3 algorithm on GPU
//
//  Created by gangliao on 12/22/14.
//  Copyright (c) 2014 gangliao. All rights reserved.
//

#include "head.h"



void read_data(char *filename, char *buffer, int num){
	FILE *fh;
	fh = fopen(filename, "r");
	fread(buffer, 1, num, fh);
	buffer[num] = '\0';
	fclose(fh);
}



int main(int argc, char* argv[])
{
	//freopen("data","r",stdin);
	//freopen("output.txt","w",stdout);

	//clock_t start, end;						    //record time
	//double runTime;
	float milliseconds = 0;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	char* filename = "genome.txt";				//load the local data set


	int n;										//input size

	char *data;									//data set pointer
	int i = 0;									//index
	//int *inp;									//transformed data pointer
	int *SA;									//Suffix Array pointer

	printf("Please input the size of dataset you want to evaluate (10 - 1000000): \t");
	scanf("%d", &n);

	data = (char *) malloc((n+1)*sizeof(char));

	read_data(filename, data, n);				//read data set from the local file


	//inp = (int *)malloc( (n+3)*sizeof(int) );	//dynamic allocate memory
	//SA  = (int *)malloc( (n+3)*sizeof(int) );
	thrust::host_vector<int> h_inp(n+3);
	thrust::host_vector<int> h_SA(n+3, 0);
	thrust::device_vector<int>d_inp;
	thrust::device_vector<int>d_SA;


	for(i=0;i<n;i++)							//Ascii 'A' -> integer 0 by 'A' - 65
	{
		h_inp[i] = to_i(data[i]);
	}

	h_inp[i]=0;h_inp[i+1]=0;h_inp[i+2]=0;				//prepare for triples
	d_inp = h_inp;
	d_SA = h_SA;


	hipEventRecord(start);
	suffixArray(d_inp, d_SA, n, MAX_ALPHA);	        //dc3/skew algorithm

	h_SA = d_SA;
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	for(i = 0 ; i < n ; i++)					//print sorted suffixes from data set
	{
		printf("No.%d Index.", i);
		print_suffix(data, h_SA[i]);
	}

	printf("GPU construct Suffix Array\nNUM: %d \t Time: %f Sec\n", n, milliseconds/1000);

	hipEventDestroy( start );
	hipEventDestroy( stop );
	free(data);									//free allocated memory
	return 0;
}




void suffixArray(thrust::device_vector<int>& s, thrust::device_vector<int>& SA, int n, int K) {

	int n0=(n+2)/3, n1= (n+1)/3, n2=n/3, n02=n0+n2;

	thrust::device_vector<int>d_s12(n02+3, 0);
	thrust::device_vector<int>d_SA12(n02+3, 0);
	thrust::device_vector<int>d_s0(n0, 0);
	thrust::device_vector<int>d_SA0(n0, 0);
	thrust::device_vector<int>d_scan(n02+3);

	//std::cout << "d_s12 size: " << d_s12.size() << std::endl;
	// S12 initialization:
	thrust::sequence(d_s12.begin(), d_s12.begin()+n02);
	thrust::transform(d_s12.begin(), d_s12.begin()+n02, d_s12.begin(), mapping());


	dim3 numThreads(1024,1,1);
	dim3 numBlocks((n02-1)/1024 + 1,1,1);

	int *pd_s12 = thrust::raw_pointer_cast( &d_s12[0] );
	int *pd_SA12 = thrust::raw_pointer_cast( &d_SA12[0] );
	int *pd_s = thrust::raw_pointer_cast( &s[0] );
	int *pd_s0 = thrust::raw_pointer_cast( &d_s0[0] );
	int *pd_SA0 = thrust::raw_pointer_cast( &d_SA0[0] );
	int *pd_SA = thrust::raw_pointer_cast( &SA[0] );
	//radix sort - using SA12 to store keys
	keybits<<<numBlocks, numThreads>>>(pd_SA12, pd_s12, pd_s , n02, 2);
	//hipDeviceSynchronize();

	thrust::sort_by_key(d_SA12.begin(), d_SA12.begin()+n02, d_s12.begin());

	keybits<<<numBlocks, numThreads>>>(pd_SA12, pd_s12, pd_s , n02, 1);
	thrust::sort_by_key(d_SA12.begin(), d_SA12.begin()+n02, d_s12.begin());

	keybits<<<numBlocks, numThreads>>>(pd_SA12, pd_s12, pd_s , n02, 0);
	thrust::sort_by_key(d_SA12.begin(), d_SA12.begin()+n02, d_s12.begin());

	d_SA12 = d_s12;


	// stably sort the mod 0 suffixes from SA12 by their first character
	// find lexicographic names of triples
	int *pd_scan = thrust::raw_pointer_cast( &d_scan[0] );
	InitScan<<<numBlocks, numThreads>>>(pd_s, pd_SA12, pd_scan, n02);

	thrust::exclusive_scan(d_scan.begin(), d_scan.end(), d_scan.begin());


	Set_suffix_rank<<<numBlocks, numThreads>>>(pd_s12, pd_SA12, pd_scan, n02, n0);

	//for(int i = 0; i < d_s12.size(); i++)
	//							std::cout << "s12[" << i << "] = " << d_s12[i]<< std::endl;

	int max_rank = d_scan[n02];
	//std::cout << max_rank << std::endl;
	//int max_rank = set_suffix_rank(s,s12,SA12,n02,n0);


	// if max_rank is less than the size of s12, we have a repeat. repeat dc3.
	// else generate the suffix array of s12 directly

	if(max_rank < n02)
	{
		suffixArray(d_s12,d_SA12,n02,max_rank);
		Store_unique_ranks<<<numBlocks, numThreads>>>(pd_s12, pd_SA12, n02);
	}else{
		Compute_SA_From_UniqueRank<<<numBlocks, numThreads>>>(pd_s12, pd_SA12, n02);
	}




	InitScan2<<<numBlocks, numThreads>>>(pd_SA12, pd_scan, n0, n02);
	thrust::exclusive_scan(d_scan.begin(), d_scan.end(), d_scan.begin());
	Set_S0<<<numBlocks, numThreads>>>(pd_s0, pd_SA12, pd_scan, n0, n02);


	dim3 numBlocks3((n0-1)/1024 + 1);
	keybits<<<numBlocks3, numThreads>>>(pd_SA0, pd_s0, pd_s, n0, 0);
	thrust::sort_by_key(d_SA0.begin(), d_SA0.begin() + n0, d_s0.begin());
	d_SA0 = d_s0;

	for(int i = 0; i < d_SA0.size(); i++)
					std::cout << "SA0[" << i << "] = " << d_SA0[i]<< std::endl;

	for(int i = 0; i < d_SA12.size(); i++)
					std::cout << "SA12[" << i << "] = " << d_SA12[i]<< std::endl;



	// merge sorted SA0 suffixes and sorted SA12 suffixes
	dim3 numBlocks2((n-1)/1024 + 1);
	merge_suffixes<<<numBlocks, numThreads>>>(pd_SA0, pd_SA12, pd_SA, pd_s, pd_s12, n0, n02, n);
	hipDeviceSynchronize();
	///for(int i = 0; i < SA.size(); i++)
	//	std::cout << "SA[" << i << "] = " << SA[i]<< std::endl;

	for(int i = 0; i < s.size(); i++)
		std::cout << "s[" << i << "] = " << (char)s[i]<< std::endl;



}

